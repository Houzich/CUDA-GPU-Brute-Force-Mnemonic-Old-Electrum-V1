﻿/**
  ******************************************************************************
  * @author		Anton Houzich
  * @version	V1.0.0
  * @date		20-March-2023
  * @mail		houzich_anton@mail.ru
  * discussion  https://t.me/BRUTE_FORCE_CRYPTO_WALLET
  ******************************************************************************
  */


#include <stdafx.h>

#include <iostream>
#include <chrono>
#include <thread>
#include <fstream>
#include <string>
#include <memory>
#include <sstream>
#include <iomanip>
#include <vector>
#include <map>
#include <omp.h>



#include "Dispatcher.h"
#include "GPU.h"
#include "KernelStride.hpp"
#include "Helper.h"


#include "hip/hip_runtime.h"
#include ""


#include "../Tools/tools.h"
#include "../Tools/utils.h"
#include "../config/Config.hpp"
#include "../Tools/segwit_addr.h"



uint64_t number_of_addresses_generate = 0;
int num_bytes = 0;




int Generate_Mnemonic_And_Hash(void)
{
	hipError_t cudaStatus = hipSuccess;

	ConfigClass config;
	try {
		parse_config(&config, "config.cfg");
	}
	catch (...) {
		for (;;)
			std::this_thread::sleep_for(std::chrono::seconds(30));
	}


	devicesInfo();
	// Choose which GPU to run on, change this on a multi-GPU system.
	uint32_t num_device = 0;
#ifndef TEST_MODE
	std::cout << "\n\nEnter number of device: ";
	std::cin >> num_device;
#endif //GENERATE_INFINITY
	cudaStatus = hipSetDevice(num_device);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return -1;
	}

	size_t num_wallets_gpu = config.cuda_grid * config.cuda_block;

	tools::clearFiles();
	//18,446,744,073,709,551,615
	uint64_t number_of_addresses = 0;
	int count_save_data_in_file = 0;

	std::cout << "\nNUM WALLETS IN ROUND GPU: " << tools::formatWithCommas(num_wallets_gpu) << std::endl << std::endl;
#ifndef TEST_MODE
	std::cout << "Max value: 18,000,000,000,000,000,000 (18000000000000000000)" << std::endl;
	std::cout << "Enter number of generate mnemonic: ";
	std::cin >> number_of_addresses;
	number_of_addresses = (((number_of_addresses - 1) / (num_wallets_gpu)+1) * (num_wallets_gpu));

	std::cout << "Enter num rounds save data in file: ";
	std::cin >> count_save_data_in_file;

	std::cout << "!!!FOR TEST!!! Enter num bytes for check 6...8: ";
	std::cin >> num_bytes;
	if (num_bytes != 0)
		if ((num_bytes < 6) || (num_bytes > 8)) {
			std::cout << "Error num bytes. Won't be used!" << std::endl;
			num_bytes = 0;
		}


#else
	number_of_addresses = num_wallets_gpu*1;
	num_bytes = 5;
	count_save_data_in_file = 1;
#endif //TEST_MODE
	data_class* Board = new data_class();
	stride_class* Stride = new stride_class(Board);

	int err = tools::readAllTables(Board->host.tables, config.folder_database, "");
	if (err == -1) {
		std::cout << "Error get_all_tables segwit!" << std::endl;
		goto Error;
	}


	if (Board->malloc(config.cuda_grid, config.cuda_block, count_save_data_in_file == 0 ? false : true) != 0) {
		std::cout << "Error Board->Malloc()!" << std::endl;
		goto Error;
	}

	if (Stride->init() != 0) {
		printf("Error INIT!!\n");
		goto Error;
	}

	Board->host.freeTableBuffers();

	std::cout << "START GENERATE ADDRESSES!" << std::endl;
	std::cout << "\nGENERATE " << tools::formatWithCommas(number_of_addresses) << " MNEMONICS. " << tools::formatWithCommas(number_of_addresses * NUM_ALL_CHILDS) << " ADDRESSES. PACKET " << tools::formatWithCommas(Board->wallets_in_round_gpu) << ". WAIT...\n\n";

	tools::generateRandomWordsIndex(Board->host.words_index, WORDS_MNEMONIC);

	if (hipMemcpyToSymbol(HIP_SYMBOL(num_bytes_find), &num_bytes, 4, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		fprintf(stderr, "hipMemcpyToSymbol to num_bytes_find failed!");
		goto Error;
	}


	static int start_save = 0;
	for (uint64_t step = 0; step < number_of_addresses / (Board->wallets_in_round_gpu); step++)
	{
		tools::start_time();

		number_of_addresses_generate = (step + 1) * (Board->wallets_in_round_gpu);
		if (start_save < count_save_data_in_file) {
			if (Stride->start_for_save(config.cuda_grid, config.cuda_block) != 0) {
				printf("Error START!!\n");
				goto Error;
			}
		}
		else
		{
			if (Stride->start(config.cuda_grid, config.cuda_block) != 0) {
				printf("Error START!!\n");
				goto Error;
			}
		}


		tools::generateRandomWordsIndex(Board->host.words_index, WORDS_MNEMONIC);
		if (start_save < count_save_data_in_file) {
			if (Stride->end_for_save() != 0) {
				printf("Error END!!\n");
				goto Error;
			}
		}
		else
		{
			if (Stride->end() != 0) {
				printf("Error END!!\n");
				goto Error;
			}
		}

		if (start_save < count_save_data_in_file) {
			start_save++;
			tools::saveResult((char*)Board->host.mnemonic, (uint8_t*)Board->host.hash160, Board->wallets_in_round_gpu);
		}

		tools::checkResult(Board->host.ret);

		float delay;
		tools::stop_time_and_calc(&delay);
		std::cout << "\rSPEED: " << std::setw(8) << std::fixed << tools::formatWithCommas((float)Board->wallets_in_round_gpu / (delay / 1000.0f)) << " MNEMONICS/SECOND AND "
			<< tools::formatWithCommas(((float)Board->wallets_in_round_gpu * NUM_ALL_CHILDS) / (delay / 1000.0f)) << " ADDRESSES/SECOND, ROUND: " << step;
	}

	std::cout << "\n\nEND!" << std::endl;

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}

	return 0;
Error:
	std::cout << "\n\nERROR!" << std::endl;
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}

	return -1;
}







