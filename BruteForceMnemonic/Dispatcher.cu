﻿/**
  ******************************************************************************
  * @author		Anton Houzich
  * @version	V2.0.0
  * @date		9-May-2023
  * @mail		houzich_anton@mail.ru
  * discussion  https://t.me/BRUTE_FORCE_CRYPTO_WALLET
  ******************************************************************************
  */


#include <stdafx.h>

#include <iostream>
#include <chrono>
#include <thread>
#include <fstream>
#include <string>
#include <memory>
#include <sstream>
#include <iomanip>
#include <vector>
#include <map>
#include <omp.h>



#include "Dispatcher.h"
#include "GPU.h"
#include "KernelStride.hpp"
#include "Helper.h"


#include "hip/hip_runtime.h"
#include ""


#include "../Tools/tools.h"
#include "../Tools/utils.h"
#include "../config/Config.hpp"
#include "../Tools/segwit_addr.h"



static std::thread save_thread;

int Generate_Mnemonic_And_Hash(void)
{
	hipError_t cudaStatus = hipSuccess;
	int err;
	ConfigClass Config;
	try {
		parse_config(&Config, "config.cfg");
		err = tools::stringToWordIndices(Config.static_words_generate_mnemonic, Config.words_indicies_mnemonic);
		if (err != 0)
		{
			std::cerr << "Error stringToWordIndices()!" << std::endl;
			return -1;
		}
		uint64_t number_of_generated_mnemonics = (Config.number_of_generated_mnemonics / (Config.cuda_block * Config.cuda_grid)) * (Config.cuda_block * Config.cuda_grid);
		if ((Config.number_of_generated_mnemonics % (Config.cuda_block * Config.cuda_grid)) != 0) number_of_generated_mnemonics += Config.cuda_block * Config.cuda_grid;
		Config.number_of_generated_mnemonics = number_of_generated_mnemonics;
	}
	catch (...) {
		for (;;)
			std::this_thread::sleep_for(std::chrono::seconds(30));
	}


	devicesInfo();
	// Choose which GPU to run on, change this on a multi-GPU system.
	uint32_t num_device = 0;
#ifndef TEST_MODE
	std::cout << "\n\nEnter number of device: ";
	std::cin >> num_device;
#endif //GENERATE_INFINITY
	cudaStatus = hipSetDevice(num_device);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return -1;
	}

	size_t num_wallets_gpu = Config.cuda_grid * Config.cuda_block;
	if (num_wallets_gpu < NUM_PACKETS_SAVE_IN_FILE)
	{
		std::cerr << "Error num_wallets_gpu < NUM_PACKETS_SAVE_IN_FILE!" << std::endl;
		return -1;
	}

	uint32_t num_bytes = 0;
	if (Config.chech_equal_bytes_in_adresses == "yes")
	{
#ifdef TEST_MODE
		num_bytes = 6;
#else
		num_bytes = 8;
#endif //TEST_MODE
	}

	std::cout << "\nNUM WALLETS IN ROUND GPU: " << tools::formatWithCommas(num_wallets_gpu) << std::endl << std::endl;

	data_class* Data = new data_class();
	stride_class* Stride = new stride_class(Data);
	size_t num_addresses_in_tables = 0;
	std::cout << "READ TABLES! WAIT..." << std::endl;
	tools::clearFiles();
	err = tools::readAllTables(Data->host.tables, Config.folder_tables, "", &num_addresses_in_tables);
	if (err == -1) {
		std::cerr << "Error get_all_tables segwit!" << std::endl;
		goto Error;
	}

	if (num_addresses_in_tables == 0) {
		std::cerr << "ERROR READ TABLES!! NO ADDRESSES IN FILES!!" << std::endl;
		goto Error;
	}

	if (Data->malloc(Config.cuda_grid, Config.cuda_block, Config.num_paths, Config.num_child_addresses, Config.save_generation_result_in_file == "yes" ? true : false) != 0) {
		std::cerr << "Error Data->Malloc()!" << std::endl;
		goto Error;
	}

	if (Stride->init() != 0) {
		std::cerr << "Error INIT!!" << std::endl;
		goto Error;
	}

	Data->host.freeTableBuffers();

	std::cout << "START GENERATE ADDRESSES!" << std::endl;
	std::cout << "\nGENERATE " << tools::formatWithCommas(Config.number_of_generated_mnemonics) << " MNEMONICS. " << tools::formatWithCommas(Config.number_of_generated_mnemonics * Data->num_all_childs) << " ADDRESSES. MNEMONICS IN ROUNDS " << tools::formatWithCommas(Data->wallets_in_round_gpu) << ". WAIT...\n\n";



	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_num_bytes_find), &num_bytes, 4, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol to num_bytes_find failed!" << std::endl;
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_generate_path), &Config.generate_path, sizeof(Config.generate_path), 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol to dev_generate_path failed!" << std::endl;
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_num_childs), &Config.num_child_addresses, 4, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol to dev_num_child failed!" << std::endl;
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_num_paths), &Config.num_paths, 4, 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol to dev_num_paths failed!" << std::endl;
		goto Error;
	}
	if (hipMemcpyToSymbol(HIP_SYMBOL(dev_static_words_indices), &Config.words_indicies_mnemonic, NUM_WORDS_MNEMONIC * sizeof(uint16_t), 0, hipMemcpyHostToDevice) != hipSuccess)
	{
		std::cerr << "hipMemcpyToSymbol to dev_gen_words_indices failed!" << std::endl;
		goto Error;
	}

	tools::generateRandomWordsIndex(Data->host.words_index, Data->size_words_index_buf / sizeof(uint16_t));

	for (uint64_t step = 0; step < Config.number_of_generated_mnemonics / (Data->wallets_in_round_gpu); step++)
	{
		tools::start_time();

		if (Config.save_generation_result_in_file == "yes") {
			if (Stride->start_for_save(Config.cuda_grid, Config.cuda_block) != 0) {
				std::cerr << "Error START!!" << std::endl;
				goto Error;
			}
		}
		else
		{
			if (Stride->start(Config.cuda_grid, Config.cuda_block) != 0) {
				std::cerr << "Error START!!" << std::endl;
				goto Error;
			}
		}


		tools::generateRandomWordsIndex(Data->host.words_index, Data->size_words_index_buf / sizeof(uint16_t));

		if (save_thread.joinable()) save_thread.join();

		if (Config.save_generation_result_in_file == "yes") {
			if (Stride->end_for_save() != 0) {
				std::cerr << "Error END!!" << std::endl;
				goto Error;
			}
		}
		else
		{
			if (Stride->end() != 0) {
				std::cerr << "Error END!!" << std::endl;
				goto Error;
			}
		}

		if (Config.save_generation_result_in_file == "yes") {
			save_thread = std::thread(&tools::saveResult, (char*)Data->host.mnemonic, (uint8_t*)Data->host.hash160, Data->wallets_in_round_gpu, Data->num_all_childs);
		}

		tools::checkResult(Data->host.ret);



		double delay;
		tools::stop_time_and_calc_sec(&delay);
		std::cout << "\rGENERATE: " << tools::formatWithCommas((double)Data->wallets_in_round_gpu / delay) << " MNEMONICS/SEC AND "
			<< tools::formatWithCommas((double)(Data->wallets_in_round_gpu * Data->num_all_childs) / delay) << " ADDRESSES/SEC"
			<< " | SCAN: " << tools::formatPrefix((double)(Data->wallets_in_round_gpu * Data->num_all_childs * num_addresses_in_tables) / delay) << " ADDRESSES/SEC"
			<< " | ROUND: " << step;
	}

	std::cout << "\n\nEND!" << std::endl;
	if (save_thread.joinable()) save_thread.join();
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}

	return 0;
Error:
	std::cout << "\n\nERROR!" << std::endl;
	if (save_thread.joinable()) save_thread.join();
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}

	return -1;
}







