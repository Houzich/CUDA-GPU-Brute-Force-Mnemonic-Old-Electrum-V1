#include "hip/hip_runtime.h"
﻿/**
  ******************************************************************************
  * @author		Anton Houzich
  * @version	V2.0.0
  * @date		9-May-2023
  * @mail		houzich_anton@mail.ru
  * discussion  https://t.me/BRUTE_FORCE_CRYPTO_WALLET
  ******************************************************************************
  */

#include <stdafx.h>
#include "hip/hip_runtime.h"
#include ""

#include "Helper.h"




hipError_t deviceSynchronize(std::string name_kernel) {
	hipError_t cudaStatus = hipSuccess;
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipGetLastError \"%s\" launch failed: %s\n", name_kernel.c_str(), hipGetErrorString(cudaStatus));
		return cudaStatus;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize \"%s\" returned error code \"%s\" after launching addKernel!\n", name_kernel.c_str(), hipGetErrorString(cudaStatus));
		return cudaStatus;
	}
	return cudaStatus;
}

// Beginning of GPU Architecture definitions
inline int _ConvertSMVer2Cores(int major, int minor)
{
	// Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
	typedef struct
	{
		int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] =
	{
		{ 0x10,  8 }, // Tesla Generation (SM 1.0) G80 class
		{ 0x11,  8 }, // Tesla Generation (SM 1.1) G8x class
		{ 0x12,  8 }, // Tesla Generation (SM 1.2) G9x class
		{ 0x13,  8 }, // Tesla Generation (SM 1.3) GT200 class
		{ 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
		{ 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
		{ 0x30, 192}, // Kepler Generation (SM 3.0) GK10x class
		{ 0x35, 192}, // Kepler Generation (SM 3.5) GK11x class
		{   -1, -1 }
	};

	int index = 0;

	while (nGpuArchCoresPerSM[index].SM != -1)
	{
		if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
		{
			return nGpuArchCoresPerSM[index].Cores;
		}

		index++;
	}

	// If we don't find the values, we default use the previous one to run properly
	printf("MapSMtoCores for SM %d.%d is undefined.  Default to use %d Cores/SM\n", major, minor, nGpuArchCoresPerSM[7].Cores);
	return nGpuArchCoresPerSM[7].Cores;
}
// end of GPU Architecture definitions


void devicesInfo(void)
{
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);

	if (deviceCount == 0)
	{
		printf("\nThere are no available device(s) that support CUDA\n");
	}
	else
	{
		printf("\nDetected %d CUDA Capable device(s)\n", deviceCount);
	}

	int dev = 0, driverVersion = 0, runtimeVersion = 0;
	for (dev = 0; dev < deviceCount; ++dev)
	{
		hipSetDevice(dev);
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);

		printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

		// Console log
		hipDriverGetVersion(&driverVersion);
		hipRuntimeGetVersion(&runtimeVersion);
		printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n", driverVersion / 1000, (driverVersion % 100) / 10, runtimeVersion / 1000, (runtimeVersion % 100) / 10);
		//printf("  CUDA Capability Major/Minor version number:    %d.%d\n", deviceProp.major, deviceProp.minor);

		char msg[256];
		sprintf(msg, "  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
			(float)deviceProp.totalGlobalMem / 1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);
		printf("%s", msg);

		//printf("  (%2d) Multiprocessors, (%3d) CUDA Cores/MP:     %d CUDA Cores\n",
		//	deviceProp.multiProcessorCount,
		//	_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
		//	_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount);
		printf("  GPU Max Clock rate:                            %.0f MHz (%0.2f GHz)\n", (float)deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);

//
//#if CUDART_VERSION >= 5000
//		// This is supported in CUDA 5.0 (runtime API device properties)
//		printf("  Memory Clock rate:                             %.0f Mhz\n", deviceProp.memoryClockRate * 1e-3f);
//		printf("  Memory Bus Width:                              %d-bit\n", deviceProp.memoryBusWidth);
//
//		if (deviceProp.l2CacheSize)
//		{
//			printf("  L2 Cache Size:                                 %d bytes\n", deviceProp.l2CacheSize);
//		}
//
//#else
//		// This only available in CUDA 4.0-4.2 (but these were only exposed in the CUDA Driver API)
//		int memoryClock;
//		getCudaAttribute<int>(&memoryClock, hipDeviceAttributeMemoryClockRate, dev);
//		printf("  Memory Clock rate:                             %.0f Mhz\n", memoryClock * 1e-3f);
//		int memBusWidth;
//		getCudaAttribute<int>(&memBusWidth, hipDeviceAttributeMemoryBusWidth, dev);
//		printf("  Memory Bus Width:                              %d-bit\n", memBusWidth);
//		int L2CacheSize;
//		getCudaAttribute<int>(&L2CacheSize, hipDeviceAttributeL2CacheSize, dev);
//
//		if (L2CacheSize)
//		{
//			printf("  L2 Cache Size:                                 %d bytes\n", L2CacheSize);
//		}
//
//#endif
//
//		printf("  Maximum Texture Dimension Size (x,y,z)         1D=(%d), 2D=(%d, %d), 3D=(%d, %d, %d)\n",
//			deviceProp.maxTexture1D, deviceProp.maxTexture2D[0], deviceProp.maxTexture2D[1],
//			deviceProp.maxTexture3D[0], deviceProp.maxTexture3D[1], deviceProp.maxTexture3D[2]);
//		printf("  Maximum Layered 1D Texture Size, (num) layers  1D=(%d), %d layers\n",
//			deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1]);
//		printf("  Maximum Layered 2D Texture Size, (num) layers  2D=(%d, %d), %d layers\n",
//			deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1], deviceProp.maxTexture2DLayered[2]);
//
//
//		printf("  Total amount of constant memory:               %llu bytes\n", (uint64_t)deviceProp.totalConstMem);
//		printf("  Total amount of shared memory per block:       %llu bytes\n", (uint64_t)deviceProp.sharedMemPerBlock);
//		printf("  Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
//		printf("  Warp size:                                     %d\n", deviceProp.warpSize);
//		printf("  Maximum number of threads per multiprocessor:  %d\n", deviceProp.maxThreadsPerMultiProcessor);
//		printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
//		printf("  Max dimension size of a thread block (x,y,z): (%d, %d, %d)\n",
//			deviceProp.maxThreadsDim[0],
//			deviceProp.maxThreadsDim[1],
//			deviceProp.maxThreadsDim[2]);
//		printf("  Max dimension size of a grid size    (x,y,z): (%d, %d, %d)\n",
//			deviceProp.maxGridSize[0],
//			deviceProp.maxGridSize[1],
//			deviceProp.maxGridSize[2]);
//		printf("  Maximum memory pitch:                          %llu bytes\n", (uint64_t)deviceProp.memPitch);
//		printf("  Texture alignment:                             %llu bytes\n", (uint64_t)deviceProp.textureAlignment);
//		printf("  Concurrent copy and kernel execution:          %s with %d copy engine(s)\n", (deviceProp.deviceOverlap ? "Yes" : "No"), deviceProp.asyncEngineCount);
//		printf("  Run time limit on kernels:                     %s\n", deviceProp.kernelExecTimeoutEnabled ? "Yes" : "No");
//		printf("  Integrated GPU sharing Host Memory:            %s\n", deviceProp.integrated ? "Yes" : "No");
//		printf("  Support host page-locked memory mapping:       %s\n", deviceProp.canMapHostMemory ? "Yes" : "No");
//		printf("  Alignment requirement for Surfaces:            %s\n", deviceProp.surfaceAlignment ? "Yes" : "No");
//		printf("  Device has ECC support:                        %s\n", deviceProp.ECCEnabled ? "Enabled" : "Disabled");
//#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
//		printf("  CUDA Device Driver Mode (TCC or WDDM):         %s\n", deviceProp.tccDriver ? "TCC (Tesla Compute Cluster Driver)" : "WDDM (Windows Display Driver Model)");
//#endif
//		printf("  Device supports Unified Addressing (UVA):      %s\n", deviceProp.unifiedAddressing ? "Yes" : "No");
//		printf("  Device PCI Domain ID / Bus ID / location ID:   %d / %d / %d\n", deviceProp.pciDomainID, deviceProp.pciBusID, deviceProp.pciDeviceID);

		//const char* sComputeMode[] =
		//{
		//	"Default (multiple host threads can use ::hipSetDevice() with device simultaneously)",
		//	"Exclusive (only one host thread in one process is able to use ::hipSetDevice() with this device)",
		//	"Prohibited (no host thread can use ::hipSetDevice() with this device)",
		//	"Exclusive Process (many threads in one process is able to use ::hipSetDevice() with this device)",
		//	"Unknown",
		//	NULL
		//};
		//printf("  Compute Mode:\n");
		//printf("     < %s >\n", sComputeMode[deviceProp.computeMode]);
	}
}





