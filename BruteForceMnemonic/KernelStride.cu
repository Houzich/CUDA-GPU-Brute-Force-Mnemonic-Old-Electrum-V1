﻿/**
  ******************************************************************************
  * @author		Anton Houzich
  * @version	V2.0.0
  * @date		9-May-2023
  * @mail		houzich_anton@mail.ru
  * discussion  https://t.me/BRUTE_FORCE_CRYPTO_WALLET
  ******************************************************************************
  */
#include <stdafx.h>
#include <stdio.h>
#include <stdint.h>


#include "KernelStride.hpp"
#include "Helper.h"
#include <GPU.h>
#include "../Tools/utils.h"


int stride_class::bruteforce_mnemonic(uint64_t grid, uint64_t block) {
	gl_bruteforce_mnemonic << <(uint32_t)grid, (uint32_t)block, 0, dt->stream1 >> > (dt->dev.words_index, dt->dev.dev_tables, dt->dev.ret);
	return 0;
}

int stride_class::bruteforce_mnemonic_for_save(uint64_t grid, uint64_t block) {
	gl_bruteforce_mnemonic_for_save << <(uint32_t)grid, (uint32_t)block, 0, dt->stream1 >> > (dt->dev.words_index, dt->dev.dev_tables, dt->dev.ret, dt->dev.mnemonic, dt->dev.hash160);
	return 0;
}

int stride_class::memsetGlobalMnemonic()
{
	if (hipMemcpyAsync(dt->dev.words_index, dt->host.words_index, dt->size_words_index_buf, hipMemcpyHostToDevice, dt->stream1) != hipSuccess) { fprintf(stderr, "hipMemcpyAsync to Board->dev.words_index failed!"); return -1; }
	if (hipMemsetAsync(dt->dev.ret, 0, sizeof(retStruct), dt->stream1) != hipSuccess) { fprintf(stderr, "hipMemset Board->dev.ret failed!"); return -1; }
	return 0;
}

int stride_class::memsetGlobalMnemonicSave()
{
	if (hipMemcpyAsync(dt->dev.words_index, dt->host.words_index, dt->size_words_index_buf, hipMemcpyHostToDevice, dt->stream1) != hipSuccess) { fprintf(stderr, "hipMemcpyAsync to Board->dev.words_index failed!"); return -1; }
	if (hipMemsetAsync(dt->dev.ret, 0, sizeof(retStruct), dt->stream1) != hipSuccess) { fprintf(stderr, "hipMemset Board->dev.ret failed!"); return -1; }
	return 0;
}

int stride_class::cudaMallocDevice(uint8_t** point, uint64_t size, uint64_t* all_gpu_memory_size, std::string buff_name) {
	if (hipMalloc(point, size) != hipSuccess) {
		fprintf(stderr, "hipMalloc (%s) failed! Size: %s", buff_name.c_str(), tools::formatWithCommas(size).data()); return -1;
	}
	*all_gpu_memory_size += size;
	if(size == 0)
		std::cout << "!!! WORNING !!! MALLOC GPU MEMORY SIZE (" << buff_name << "): 0.000000 MB\n";
	else
		std::cout << "MALLOC GPU MEMORY SIZE (" << buff_name << "): " << std::to_string((float)size / (1024.0f * 1024.0f)) << " MB\r";
	return 0;
}

int stride_class::init()
{
	size_t memory_size = 0;
	for (int i = 0; i < 256; i++)
	{
		std::string name = "Table " + tools::byteToHexString(i);
		if (cudaMallocDevice((uint8_t**)&dt->dev.tables[i].table, dt->host.tables[i].size, &memory_size, name.c_str()) != 0)
		{
			std::cout << "Error cudaMallocDevice(), Board->dev.table_legacy[i]! i = " << i << std::endl;
			return -1;
		}
		dt->dev.tables[i].size = dt->host.tables[i].size;
		dt->dev.memory_size += dt->host.tables[i].size;
	}
	std::cout << "MALLOC MEMORY SIZE (TABLES GPU): " << std::to_string((float)memory_size / (1024.0f * 1024.0f)) << " MB\n";

	std::cout << "INIT GPU ... \n";
	for (int i = 0; i < 256; i++)
	{
		if (hipMemcpy((void*)dt->dev.tables[i].table, dt->host.tables[i].table, dt->host.tables[i].size, hipMemcpyHostToDevice) != hipSuccess)
		{
			std::cout << "hipMemcpy to Board->dev.table_legacy[i] failed! i = " << i << std::endl;
			return -1;
		}
		const size_t percentDone = (i * 100 / 256) / 2;
		std::cout << "  " << percentDone << "%\r";
	}
	if (hipMemcpy(dt->dev.dev_tables, dt->dev.tables, 256 * sizeof(tableStruct), hipMemcpyHostToDevice) != hipSuccess) { fprintf(stderr, "hipMemcpyAsync to Board->dev.table_legacy failed!"); return -1; }
	if (deviceSynchronize("init") != hipSuccess) return -1;
	return 0;
}

int stride_class::start_for_save(uint64_t grid, uint64_t block)
{
	if (memsetGlobalMnemonicSave() != 0) return -1;
	if (bruteforce_mnemonic_for_save(grid, block) != 0) return -1;

	return 0;
}

int stride_class::start(uint64_t grid, uint64_t block)
{
	if (memsetGlobalMnemonic() != 0) return -1;
	if (bruteforce_mnemonic(grid, block) != 0) return -1;

	return 0;
}

int stride_class::end()
{
	hipError_t cudaStatus = hipSuccess;
	if (deviceSynchronize("end") != hipSuccess) return -1; //????
	cudaStatus = hipMemcpy(dt->host.ret, dt->dev.ret, sizeof(retStruct), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy ret failed!");
		return -1;
	}

	return 0;
}

int stride_class::end_for_save()
{
	hipError_t cudaStatus = hipSuccess;


	if (deviceSynchronize("end_for_save") != hipSuccess) return -1; //????
	cudaStatus = hipMemcpy(dt->host.mnemonic, dt->dev.mnemonic, dt->size_mnemonic_buf, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy mnemonic failed!");
		return -1;
	}
	cudaStatus = hipMemcpy(dt->host.hash160, dt->dev.hash160, dt->size_hash160_buf, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy hash160 failed!");
		return -1;
	}
	cudaStatus = hipMemcpy(dt->host.ret, dt->dev.ret, sizeof(retStruct), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy ret failed!");
		return -1;
	}

	return 0;
}